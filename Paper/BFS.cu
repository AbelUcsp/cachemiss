#include "hip/hip_runtime.h"
%%cu
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#define NUM_NODES 5

typedef struct
{
	int start;     // indice del primer nodo adyacente en conjunto de aristas	
	int length;    // Número de nodos adyacentes 
} Node;

__global__ void CUDA_BFS_KERNEL(Node *Va, int *Ea, bool *Fa, bool *Xa, int *Ca,bool *done)
{

	int id = threadIdx.x + blockIdx.x * blockDim.x;
	if (id > NUM_NODES)
		*done = false;


	if (Fa[id] == true && Xa[id] == false)
	{
		printf("%d ", id); //para imprimir los vértices que aparecen en el orden de BFS	
		Fa[id] = false;
		Xa[id] = true;
		__syncthreads();  //sincronización de nivel/frontera
		int k = 0;
		int i;
		int start = Va[id].start;
		int end = start + Va[id].length;
		for (int i = start; i < end; i++) 
		{
			int nid = Ea[i];

			if (Xa[nid] == false)
			{
				Ca[nid] = Ca[id] + 1;
				Fa[nid] = true;
				*done = false;
			}

		}

	}

}

//La frontera BFS corresponde a todos los nodos que se procesan en el nivel actual.


int main()
{



 clock_t begin = clock();

	 Node node[NUM_NODES];
	
	
	//int edgesSize = 2 * NUM_NODES;
	int edges[NUM_NODES];

	node[0].start = 0;
	node[0].length = 2;

	node[1].start = 2;
	node[1].length = 1;

	node[2].start = 3;
	node[2].length = 1;

	node[3].start = 4;
	node[3].length = 1;

	node[4].start = 5;
	node[4].length = 0;

	edges[0] = 1;
	edges[1] = 2;	
	edges[2] = 4;
	edges[3] = 3;
	edges[4] = 4;

	bool frontier[NUM_NODES] = { false };
	bool visited[NUM_NODES] = { false };
	int cost[NUM_NODES] = { 0 };

	int source = 0;
	frontier[source] = true;

	Node* Va;     //conjunto de vertices
	hipMalloc((void**)&Va, sizeof(Node)*NUM_NODES);
	hipMemcpy(Va, node, sizeof(Node)*NUM_NODES, hipMemcpyHostToDevice);

	int* Ea;    //conjunto de aristas
	hipMalloc((void**)&Ea, sizeof(Node)*NUM_NODES);
	hipMemcpy(Ea, edges, sizeof(Node)*NUM_NODES, hipMemcpyHostToDevice);

	bool* Fa;   //frontera
	hipMalloc((void**)&Fa, sizeof(bool)*NUM_NODES);
	hipMemcpy(Fa, frontier, sizeof(bool)*NUM_NODES, hipMemcpyHostToDevice);

	bool* Xa;   //visited
	hipMalloc((void**)&Xa, sizeof(bool)*NUM_NODES);
	hipMemcpy(Xa, visited, sizeof(bool)*NUM_NODES, hipMemcpyHostToDevice);

	int* Ca;    //cost
	hipMalloc((void**)&Ca, sizeof(int)*NUM_NODES);
	hipMemcpy(Ca, cost, sizeof(int)*NUM_NODES, hipMemcpyHostToDevice);

	

	int num_blks = 1;
	int threads = 5;



	bool done;
	bool* d_done;
	hipMalloc((void**)&d_done, sizeof(bool));
	printf("\n\n");
	int count = 0;

	printf("Order: \n\n");
	do {
		count++;
		done = true;
		hipMemcpy(d_done, &done, sizeof(bool), hipMemcpyHostToDevice);
		CUDA_BFS_KERNEL <<<num_blks, threads >>>(Va, Ea, Fa, Xa, Ca,d_done);
		hipMemcpy(&done, d_done , sizeof(bool), hipMemcpyDeviceToHost);

	} while (!done);



	printf("Parallel code took cution.\n");

	hipMemcpy(cost, Ca, sizeof(int)*NUM_NODES, hipMemcpyDeviceToHost);
	

  clock_t end = clock();
	double elapsed_time = double(end - begin) / CLOCKS_PER_SEC;
	printf("Parallel code took %.3f sec for execution.\n", elapsed_time);


/*
	printf("\nCost: ");
	for (int i = 0; i<NUM_NODES; i++)
		printf( "%d    ", cost[i]);
	printf("\n");
 */
}