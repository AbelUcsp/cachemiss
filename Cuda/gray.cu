#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <cstdio>
#include <string>
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/opencv.hpp>
#include ""

using namespace cv;
using namespace std;


#define NUM_TREADS 524

using namespace cv;


__global__ void rgb2grayKernel(unsigned char *Pout, unsigned char *Pin, int width,
                            int height, int numChannels) {
  // compute global thread coordinates
  int row = threadIdx.y + blockIdx.y * blockDim.y;
  int col = threadIdx.x + blockIdx.x * blockDim.x;

  // linearize coordinates for data access
  int grayOffset = row * width + col;


  if ((col < width) && (row < height)) {

	int rgbOffset = greyOffset*numChannels;
	unsigned char r = Pin[rgbOffset ]; // red value for pixel
	unsigned char g = Pin[rgbOffset + 2]; // green value for pixel
	unsigned char b = Pin[rgbOffset + 3]; // blue value for pixel

	Pout[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;

  }
}

void grayScale(string fileName) {
  // read image
  Mat image;
  image = imread(fileName, CV_LOAD_IMAGE_COLOR);
  if (image.empty()) {
    cout << "Cannot read image file " << fileName;
    exit(1);
  }

  // define img params and timers
  int imageChannels = 3;
  int imageWidth = image.cols;
  int imageHeight = image.rows;
  size_t size_rgb = sizeof(unsigned char) * imageWidth * imageHeight * imageChannels;
  size_t size_gray = sizeof(unsigned char) * imageWidth * imageHeight;

  // allocate mem for host image vectors
  unsigned char *h_grayImage = (unsigned char *)malloc(size_rgb);

  // grab pointer to host rgb image
  unsigned char *h_rgbImage = image.data;

  // allocate mem for device rgb and gray
  unsigned char *d_rgbImage;
  unsigned char *d_grayImage;
  hipMalloc(&d_rgbImage, size_rgb);
  hipMalloc(&d_grayImage, size_gray);

  // copy the rgb image from the host to the device and record the needed time
  hipMemcpy(d_rgbImage, h_rgbImage, size_rgb, hipMemcpyHostToDevice);

  // execution configuration parameters + kernel launch
  dim3 dimBlock(16, 16);
  dim3 dimGrid(ceil(imageWidth / 16.0), ceil(imageHeight / 16.0));
  rgb2grayKernel<<<dimGrid, dimBlock>>>(d_grayImage, d_rgbImage, imageWidth,
                                     imageHeight, imageChannels);

  hipMemcpy(h_grayImage, d_grayImage, size_gray, hipMemcpyDeviceToHost);

  // display images
  Mat imageGray(imageHeight, imageWidth, CV_8UC1, h_grayImage);

  imwrite("./grayscale.jpg", imageGray);

  // free host and device memory
  image.release();
  imageGray.release();
  free(h_grayImage);

  hipFree(d_rgbImage);
  hipFree(d_grayImage);
}




int main(int argc, char **argv) {
		string img_path = "C:/Users/USER/Downloads/opencv/Van_Gogh.jpg";
		//Van_Gogh.jpg
		string image_path(img_path);// , cv::ImreadModes::IMREAD_GRAYSCALE);
		grayScale(image_path)
		
		return 0;
}
